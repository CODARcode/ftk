#include "hip/hip_runtime.h"
#include <nvfunctional>
#include <cstdio>
#include <ftk/numeric/inverse_linear_interpolation_solver.hh>
#include <ftk/numeric/linear_interpolation.hh>
#include <ftk/numeric/symmetric_matrix.hh>
#include <ftk/numeric/fixed_point.hh>
#include <ftk/numeric/critical_point_type.hh>
#include <ftk/numeric/critical_point_test.hh>
#include <ftk/hypermesh/lattice.hh>
#include <ftk/filters/critical_point.hh>
#include "common.cuh"

//// 
template <int scope>
__device__
bool check_simplex_cp2t(
    int current_timestep,
    const lattice3_t& domain,
    const lattice3_t& core, 
    const lattice2_t& ext, 
    const element32_t& e, 
    const double *V[2], // last and current timesteps
    const double *gradV[2], // jacobian of last and current timesteps
    const double *scalar[2],
    cp3_t &cp)
{
  typedef ftk::fixed_point<> fp_t;

  const int last_timestep = current_timestep - 1;
  if (scope == scope_interval && e.corner[2] != last_timestep)
    return false;

  int vertices[3][3], indices[3];
  for (int i = 0; i < 3; i ++) {
    for (int j = 0; j < 3; j ++) {
      vertices[i][j] = e.corner[j] 
        + unit_simplex_offset_3_2<scope>(e.type, i, j);
      if (vertices[i][j] < domain.st[j] || 
          vertices[i][j] > domain.st[j] + domain.sz[j] - 1)
        return false;
    }
    indices[i] = domain.to_index(vertices[i]);;
  }

  double v[3][2];
  fp_t vf[3][2];
  for (int i = 0; i < 3; i ++) {
    size_t k = ext.to_index(vertices[i]);
    for (int j = 0; j < 2; j ++) {
      v[i][j] = V[unit_simplex_offset_3_2<scope>(e.type, i, 2/*time dimension id*/)][k*2+j];
      vf[i][j] = v[i][j];
    }
  }

  double mu[3];
  bool succ = ftk::inverse_lerp_s2v2(v, mu, 0.0);
 
  if (succ) {
    // linear jacobian interpolation
    if (gradV[1]) { // have given jacobian
      double Js[3][2][2], J[2][2];
      for (int i = 0; i < 3; i ++) {
        size_t ii = ext.to_index(vertices[i]);
        int t = unit_simplex_offset_3_2<scope>(e.type, i, 2);
        for (int j = 0; j < 2; j ++) 
          for (int k = 0; k < 2; k ++)
            Js[i][j][k] = gradV[t][ii*4 + j*2 + k];
      }
      ftk::lerp_s2m2x2(Js, mu, J);
      ftk::make_symmetric2x2(J);
      cp.type = ftk::critical_point_type_2d(J, true/*symmetric*/);
      // if (cp.type != 0x100) return false;
    }

    // scalar interpolation
    if (scalar[1]) { // have given scalar
      double values[3];
      for (int i = 0; i < 3; i ++) {
        size_t ii = ext.to_index(vertices[i]);
        int t = unit_simplex_offset_3_2<scope>(e.type, i, 2);
        values[i] = scalar[t][ii];
      }
      cp.scalar = ftk::lerp_s2(values, mu);
      // if (abs(cp.scalar) < 0.02) return false; // threshold
    }

    // location interpolation
    double X[3][3];
    for (int i = 0; i < 3; i ++)
      for (int j = 0; j < 3; j ++)
        X[i][j] = vertices[i][j];
    ftk::lerp_s2v3(X, mu, cp.x);

    return true;
  } else 
    return false;
}

template <int scope>
__global__
void sweep_simplices(
    int current_timestep,
    const lattice3_t domain,
    const lattice3_t core,
    const lattice2_t ext, // array dimensions
    const double *Vc, // current timestep
    const double *Vl, // last timestep
    const double *Jc, 
    const double *Jl,
    const double *Sc, 
    const double *Sl,
    unsigned long long &ncps, cp3_t *cps)
{
  const double *V[2] = {Vl, Vc};
  const double *J[2] = {Jl, Jc};
  const double *S[2] = {Sl, Sc};
  
  int tid = getGlobalIdx_3D_1D();
  const element32_t e = element32_from_index<scope>(core, tid);

  cp3_t cp;
  bool succ = check_simplex_cp2t<scope>(
      current_timestep, 
      domain, core, ext, e, V, J, S, cp);

  if (succ) {
    unsigned long long i = atomicAdd(&ncps, 1ul);
    cp.tag = tid;
    cps[i] = cp;
  }
}

template<int scope>
static std::vector<cp3_t> extract_cp2dt(
    int current_timestep,
    const lattice3_t& domain,
    const lattice3_t& core, 
    const lattice2_t& ext, 
    const double *Vc, // 3D array: 2*W*H
    const double *Vl, 
    const double *Jc,
    const double *Jl,
    const double *Sc,
    const double *Sl)
{
  const size_t ntasks = core.n() * ntypes_3_2<scope>();
  const int maxGridDim = 1024;
  const int blockSize = 256;
  const int nBlocks = idivup(ntasks, blockSize);
  dim3 gridSize;
  
  if (nBlocks >= maxGridDim) 
    gridSize = dim3(idivup(nBlocks, maxGridDim), maxGridDim);
  else 
    gridSize = dim3(nBlocks);

  double *dVc, *dVl = NULL;
  hipMalloc((void**)&dVc, 2 * sizeof(double) * ext.n());
  hipMemcpy(dVc, Vc, 2 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);
  hipMalloc((void**)&dVl, 2 * sizeof(double) * ext.n());
  hipMemcpy(dVl, Vl, 2 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);

  double *dJc = NULL, *dJl = NULL;
  if (Jc) {
    hipMalloc((void**)&dJc, 4 * sizeof(double) * ext.n());
    hipMemcpy(dJc, Jc, 4 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);
  }
  if (Jl) {
    hipMalloc((void**)&dJl, 4 * sizeof(double) * ext.n());
    hipMemcpy(dJl, Jl, 4 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);
  }

  double *dSc = NULL, *dSl = NULL;
  if (Sc) {
    hipMalloc((void**)&dSc, sizeof(double) * ext.n());
    hipMemcpy(dSc, Sc, sizeof(double) * ext.n(), hipMemcpyHostToDevice);
  }
  if (Sl) {
    hipMalloc((void**)&dSl, sizeof(double) * ext.n());
    hipMemcpy(dSl, Sl, sizeof(double) * ext.n(), hipMemcpyHostToDevice);
  }

  unsigned long long *dncps; // number of cps
  hipMalloc((void**)&dncps, sizeof(unsigned long long));
  hipMemset(dncps, 0, sizeof(unsigned long long));

  cp3_t *dcps;
  hipMalloc((void**)&dcps, sizeof(cp3_t) * ext.n());
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMalloc/hipMemcpy");

  fprintf(stderr, "calling kernel func...\n");
  sweep_simplices<scope><<<gridSize, blockSize>>>(
      current_timestep, 
      domain, core, ext, dVc, dVl, dJc, dJl, dSc, dSl,
      *dncps, dcps);
  hipDeviceSynchronize();
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices");

  unsigned long long ncps;
  hipMemcpy(&ncps, dncps, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMemcpy, ncps");
  fprintf(stderr, "ncps=%lu\n", ncps);

  std::vector<cp3_t> cps(ncps);
  hipMemcpy(cps.data(), dcps, sizeof(cp3_t) * ncps, hipMemcpyDeviceToHost);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMemcpy, dcps");
  
  hipFree(dVc);
  hipFree(dVl);
  if (dJc) hipFree(dJc);
  if (dJl) hipFree(dJl);
  if (dSc) hipFree(dSc);
  if (dSl) hipFree(dSl);
  hipFree(dncps);
  hipFree(dcps);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipFree");
 
  hipDeviceSynchronize();
  fprintf(stderr, "exitting gpu kernel, ncps=%lu\n", ncps);

  return cps;
}

std::vector<cp3_t>
extract_cp2dt_cuda(
    int scope, 
    int current_timestep,
    const ftk::lattice& domain,
    const ftk::lattice& core, 
    const ftk::lattice& ext, 
    const double *Vc, 
    const double *Vl, 
    const double *Jc, 
    const double *Jl, 
    const double *Sc,
    const double *Sl)
{
  lattice3_t D(domain);
  lattice3_t C(core);
  lattice2_t E(ext);

  // std::cerr << "domain=" << domain 
  //   << ", core=" << core << ", current_timestep=" 
  //   << current_timestep << std::endl;

  if (scope == scope_interval) 
    return extract_cp2dt<scope_interval>(current_timestep, D, C, E, Vc, Vl, Jc, Jl, Sc, Sl);
  if (scope == scope_ordinal) 
    return extract_cp2dt<scope_ordinal>(current_timestep, D, C, E, Vc, Vl, Jc, Jl, Sc, Sl);
  else // scope == 2
    return extract_cp2dt<scope_all>(current_timestep, D, C, E, Vc, Vl, Jc, Jl, Sc, Sl);
}
