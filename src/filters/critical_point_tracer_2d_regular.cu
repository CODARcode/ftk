#include "hip/hip_runtime.h"
#include <nvfunctional>
#include <cstdio>
// #include <ftk/filters/critical_point_tracker_2d.hh>
#include "threadIdx.cuh"
#include "utils.cuh"
#include <ftk/numeric/inverse_linear_interpolation_solver.hh>
#include <ftk/numeric/linear_interpolation.hh>
#include <ftk/hypermesh/lattice.hh>

template <int N=3>
struct lite_lattice_t {
  int st[N], sz[N], prod[N];

  __host__ lite_lattice_t(const ftk::lattice& L) {
    for (int i = 0; i < N; i ++) {
      st[i] = L.starts_[i];
      sz[i] = L.sizes_[i];
      prod[i] = L.prod_[i];
    }
  };

  lite_lattice_t(int st_[N], int sz_[N]) {
    for (int i = 0; i < N; i ++) {
      st[i] = st_[i];
      sz[i] = sz_[i];
      if (i == 0) prod[i] = 1;
      else prod[i] = prod[i-1] * sz[i-1];
    }
  }

  template <typename uint=size_t>
  __device__ __host__
  uint n() const {return prod[N-1] * sz[N-1];}

  template <typename uint=size_t>
  __device__ __host__
  uint to_index(const int idx1[N]) const {
    int idx[N];
    for (int j = 0; j < N; j ++)
      idx[j] = idx1[j] - st[j];

    uint i = idx[0];
    for (int j = 1; j < N; j ++)
      i += idx[j] * prod[j];

    return i;
  }

  template <typename uint=size_t>
  __device__ __host__
  void from_index(uint i, int idx[N]) const {
    for (int j = N-1; j > 0; j --) {
      idx[j] = i / prod[j];
      i -= idx[j] * prod[j];
    }
    idx[0] = i;

    for (int j = 0; j < N; j ++)
      idx[j] += st[j];
  }
};

template <typename T, int N=3>
T access(const lite_lattice_t<N>& l, const T *p, const int idx[N])
{
  size_t i = l.to_index(idx);
  return p[i];
}

template <int N=3>
struct lite_element_t {
  int corner[N], /*d,*/ type;
};

template <int N=3>
struct lite_cp_t {
  size_t idx;
  double x[N]; 
  int type;
};
  
typedef lite_lattice_t<3> lattice3_t;
typedef lite_element_t<3> element32_t;
typedef lite_cp_t<3> cp3_t;
  
__device__ __constant__ 
int ntypes_3[4] = {1, 7, 12, 6}, 
    // unit_simplices_3_3[6][4] = {0}, // TODO
    unit_simplices_3_2[12][3][3] = {
      {{0,0,0},{0,0,1},{0,1,1}},
      {{0,0,0},{0,0,1},{1,0,1}},
      {{0,0,0},{0,0,1},{1,1,1}},
      {{0,0,0},{0,1,0},{0,1,1}},
      {{0,0,0},{0,1,0},{1,1,0}},
      {{0,0,0},{0,1,0},{1,1,1}},
      {{0,0,0},{0,1,1},{1,1,1}},
      {{0,0,0},{1,0,0},{1,0,1}},
      {{0,0,0},{1,0,0},{1,1,0}},
      {{0,0,0},{1,0,0},{1,1,1}},
      {{0,0,0},{1,0,1},{1,1,1}},
      {{0,0,0},{1,1,0},{1,1,1}}
    };
    // unit_simplices_3_1[7][2] = {0};
  
template <typename uint=size_t>
__device__ __host__
element32_t element32_from_index(const lattice3_t& l, int scope, uint i) {
  element32_t e; // TODO
  
  e.type = i % 12; // m.ntypes(dim, scope);
  uint ii = i / 12; // m.ntypes(dim, scope);
  l.from_index(ii, e.corner);

  return e;
}

template <typename uint=size_t>
__device__ __host__
uint element32_to_index(const lattice3_t& l, int scope, const int idx[3]) {
  size_t i = l.to_index(idx);
  return i * 12; // m.ntypes(dim, scope);
}
  
__device__
bool check_simplex_cp2t(
    int tid,
    const lattice3_t& domain, 
    const lattice3_t& block, 
    const element32_t& e, 
    double *V, 
    cp3_t &cp)
{
  int vertices[3][3];
  for (int i = 0; i < 3; i ++)
    for (int j = 0; j < 3; j ++) {
      vertices[i][j] = e.corner[j]
        + unit_simplices_3_2[e.type][i][j];
      if (vertices[i][j] < block.st[j] || 
          vertices[i][j] > block.st[j] + block.sz[j] - 1)
        return false;
    }

  double v[3][2];
  for (int i = 0; i < 3; i ++) {
    size_t k = block.to_index(vertices[i]);
    for (int j = 0; j < 2; j ++)
      v[i][j] = V[k*2+j];
  }

  double mu[3];
  bool succ = ftk::inverse_lerp_s2v2(v, mu, 0.0);
 
  if (succ) {
    double X[3][3];
    for (int i = 0; i < 3; i ++)
      for (int j = 0; j < 3; j ++)
        X[i][j] = vertices[i][j];
    ftk::lerp_s2v3(X, mu, cp.x);
#if 0
    printf("tid=%d, mu=%f, %f, %f, v={%f, %f, %f, %f, %f, %f}, verts=(%d, %d, %d), (%d, %d, %d), (%d, %d, %d)\n",
        tid, mu[0], mu[1], mu[2], 
        v[0][0], v[0][1], v[1][0], v[1][1], v[2][0], v[2][1],
        vertices[0][0], vertices[0][1], vertices[0][2],
        vertices[1][0], vertices[1][1], vertices[1][2],
        vertices[2][0], vertices[2][1], vertices[2][2]);
#endif
    return true;
  } else 
    return false;
}

__global__
void sweep_simplices(
    const lattice3_t domain, int scope, 
    const lattice3_t block, double *V)
{
  int tid = getGlobalIdx_3D_1D();
  const element32_t e = element32_from_index(domain, scope, tid);

  cp3_t cp;
  bool succ = check_simplex_cp2t(tid, domain, block, e, V, cp);
#if 1
  if (succ && tid < 4000000)
    printf("succ, tid=%d, x=%f, %f, %f\n", tid, 
        cp.x[0], cp.x[1], cp.x[2]);
#endif
}

static void extract_cp2dt(
    const lattice3_t& domain, int scope, 
    const lattice3_t& block, double *V/* 4D array: 2*W*H*T */)
{
  const size_t ntasks = block.n() * 12; // ntypes_3[2] = 12; ntypes_3 is in device constant memory
  const int maxGridDim = 1024;
  const int blockSize = 256;
  const int nBlocks = idivup(ntasks, blockSize);
  dim3 gridSize;
  
  if (nBlocks >= maxGridDim) 
    gridSize = dim3(idivup(nBlocks, maxGridDim), maxGridDim);
  else 
    gridSize = dim3(nBlocks);

  double *dV;
  hipMalloc((void**)&dV, 2 * sizeof(double) * block.n());
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMalloc");

  hipMemcpy(dV, V, 2*sizeof(double)*block.n(), hipMemcpyHostToDevice);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMemcpy");

  fprintf(stderr, "calling kernel func...\n");
  sweep_simplices<<<gridSize, blockSize>>>(domain, scope, block, dV);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices");

  hipFree(dV);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipFree");
  
  hipDeviceSynchronize();
  fprintf(stderr, "exit.\n");
}

void extract_cp2dt(
    const ftk::lattice& domain, int scope, 
    const ftk::lattice& block, double *V)
{
  lattice3_t D(domain), B(block);
  std::cerr << domain << std::endl;
  std::cerr << block << std::endl;
  extract_cp2dt(D, scope, B, V);
}

#if 0
int main(int argc, char **argv)
{
  int st[3] = {0, 0, 0}, 
      sz[3] = {256, 256, 16};
  lattice3_t domain(st, sz);
  double *V = NULL;
  int scope = 0;

  extract_cp2dt(domain, scope, domain, V);
  return 0;
}
#endif




#if 0
struct cp2d_tracker_context {
public:
  __device__ __host__
  void element_for_3_2(
      const lattice3_t& l, int scope,
      size_t tid, // thread id
      const nvstd::function<bool(const element32_t&)> &f)
      // const nvstd::function<void()> &f)
  {
    element32_t e = element32_from_index(l, scope, tid);
    f(e);
    // f(e);
  }

};
#endif
