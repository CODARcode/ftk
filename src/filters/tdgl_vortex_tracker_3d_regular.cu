#include "hip/hip_runtime.h"
#include <nvfunctional>
#include <cstdio>
#include <cassert>
#include <chrono>
#include <ftk/numeric/inverse_linear_interpolation_solver.hh>
#include <ftk/numeric/linear_interpolation.hh>
#include <ftk/numeric/clamp.hh>
#include <ftk/numeric/symmetric_matrix.hh>
#include <ftk/numeric/fixed_point.hh>
#include <ftk/numeric/critical_point_type.hh>
#include <ftk/numeric/critical_point_test.hh>
#include <ftk/mesh/lattice.hh>
#include <ftk/io/tdgl.hh>
// #include <ftk/filters/critical_point_lite.hh>
#include "common.cuh"

template <typename T> 
__device__
T line_integral(const T X0[], const T X1[], const T A0[], const T A1[]) 
{
  T dX[3] = {X1[0] - X0[0], X1[1] - X0[1], X1[2] - X0[2]};
  T A[3] = {A0[0] + A1[0], A0[1] + A1[1], A0[2] + A1[2]};

  return 0.5 * inner_product(A, dX);
}

template <typename T, int gauge>
__device__
inline void magnetic_potential(const tdgl_metadata_t& h, T X[3], T A[3])
{
  if (m.B[1] > 0) {
    A[0] = -m.Kex;
    A[1] = X[0] * m.B[2];
    A[2] = -X[0] * m.B[1];
  } else {
    A[0] = -X[1] * m.B[2] - m.Kex;
    A[1] = 0;
    A[2] = X[1] * m.B[0];
  }
}

template <int scope>
__device__
bool check_simplex_tdgl_vortex_3dt(
    int current_timestep,
    const lattice4_t& domain, 
    const lattice4_t& core, 
    const lattice3_t& ext, // array dimension
    const element43_t& e, 
    const double *Rho[2], // current and next timesteps
    const double *Phi[2], 
    cp_t &p)
{
  if (e.corner[3] != current_timestep)
    return false;
  
  int vertices[4][4], indices[4];
  size_t local_indices[4];
  for (int i = 0; i < 4; i ++) {
    for (int j = 0; j < 4; j ++) {
      vertices[i][j] = e.corner[j]
        + unit_simplex_offset_4_3<scope>(e.type, i, j);
      if (vertices[i][j] < domain.st[j] || 
          vertices[i][j] > domain.st[j] + domain.sz[j] - 1)
        return false;
    }
    indices[i] = domain.to_index(vertices[i]);
    local_indices[i] = ext.to_index(vertices[i]);
  }
  
  double rho[3], phi[3], re[3], im[3];
  for (int i = 0; i < 3; i ++) {
    const size_t k = local_indices[i]; // k = ext.to_index(vertices[i]);
    const size_t t = unit_simplex_offset_4_3<scope>(e.type, i, 3);
      
    rho[i] = Rho[t][k];
    phi[i] = Phi[t][k];
    re[i] = rho[i] * cos(phi[i]);
    im[i] = rho[i] * sin(phi[i]);
  }
  
  // compute contour integral
  float delta[3], phase_shift = 0;
  for (int i = 0; i < 3; i ++) { // ignoring quasi periodical boundary conditions
    int j = (i+1) % 3;
    float li = line_integral(X[i], X[j], A[i], A[j]);
    delta[i] = mod2pi1( phi[j] - phi[i] - li ); // gauge transformation
    phase_shift -= delta[i];
  }

  // check contour integral
  float critera = phase_shift / (2 * M_PI);
  if (fabs(critera) < 0.5) return false; // ignoring chiralities

  // guage transformation
  float psi[3][2]; // in re/im
  for (int i = 0; i < 3; i ++) {
    if (i != 0) phi[i] = phi[i-1] + delta[i-1];
    psi[i][0] = rho[i] * cos(phi[i]);
    psi[i][1] = rho[i] * sin(phi[i]);
  }

  // locate zero
  float mu[3], // barycentric coordinates
        cond; // condition number
  inverse_lerp_s2v2(psi, mu, &cond);

  // interpolation
  float x[4];
  lerp_s2v4(X, mu, x);

  // result
  p.x[0] = x[0];
  p.x[1] = x[1];
  p.x[2] = x[2];
  p.t = x[3];
  p.cond = cond;

  return false; // WIP
}

template <int scope>
__global__
void sweep_simplices(
    int current_timestep,
    const lattice4_t domain,
    const lattice4_t core,
    const lattice3_t ext, // array dimension
    const double *rho_c, // current timestep
    const double *rho_n, // next timestep
    const double *phi_c, 
    const double *phi_n,
    unsigned long long &ncps, cp_t *cps)
{
  const double *Rho[2] = {rho_c, rho_n};
  const double *Phi[2] = {phi_c, phi_n};
  
  int tid = getGlobalIdx_3D_1D();
  const element43_t e = element43_from_index<scope>(core, tid);

  cp_t cp;
  bool succ = check_simplex_tdgl_vortex_3dt<scope>(
      current_timestep,
      domain, core, ext, e, Rho, Phi, cp);

  if (succ) {
    unsigned long long i = atomicAdd(&ncps, 1ul);
    cp.tag = tid;
    cps[i] = cp;
  }
}

template <int scope>
static std::vector<cp_t> extract_tdgl_vortex_3dt(
    int current_timestep,
    const lattice4_t& domain,
    const lattice4_t& core, 
    const lattice3_t& ext, 
    const double *rho_c,
    const double *rho_n, 
    const double *phi_c,
    const double *phi_n)
{
  auto t0 = std::chrono::high_resolution_clock::now();

  const size_t ntasks = core.n() * ntypes_4_3<scope>();
  // fprintf(stderr, "ntasks=%zu\n", ntasks);
  const int maxGridDim = 1024;
  const int blockSize = 256;
  const int nBlocks = idivup(ntasks, blockSize);
  dim3 gridSize;
  
  if (nBlocks >= maxGridDim) 
    gridSize = dim3(idivup(nBlocks, maxGridDim), maxGridDim);
  else 
    gridSize = dim3(nBlocks);

  double *drho_c = NULL, *drho_n = NULL;
  if (rho_c) {
    hipMalloc((void**)&drho_c, 3 * sizeof(double) * ext.n());
    checkLastCudaError("[FTK-CUDA] error: sweep_simplices: allocating drho_c");
    hipMemcpy(drho_c, rho_c, 3 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);
    checkLastCudaError("[FTK-CUDA] error: sweep_simplices: copying drho_c");
  }
  if (rho_n) {
    hipMalloc((void**)&drho_n, 3 * sizeof(double) * ext.n());
    checkLastCudaError("[FTK-CUDA] error: sweep_simplices: allocating drho_l");
    hipMemcpy(drho_n, rho_n, 3 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);
    checkLastCudaError("[FTK-CUDA] error: sweep_simplices: copying drho_l");
  }
  
  double *dphi_c = NULL, *dphi_n = NULL;
  if (phi_c) {
    hipMalloc((void**)&dphi_c, 9 * sizeof(double) * ext.n());
    hipMemcpy(dphi_c, phi_c, 9 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);
  }
  if (phi_n) {
    hipMalloc((void**)&dphi_n, 9 * sizeof(double) * ext.n());
    hipMemcpy(dphi_n, phi_n, 9 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);
  }
  
  unsigned long long *dncps; // number of cps
  hipMalloc((void**)&dncps, sizeof(unsigned long long));
  hipMemset(dncps, 0, sizeof(unsigned long long));
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: allocating dncps");

  cp_t *dcps;
  hipMalloc((void**)&dcps, sizeof(cp_t) * core.n());
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: allocating dcps");
  hipDeviceSynchronize();

  fprintf(stderr, "calling kernel func...\n");
  sweep_simplices<scope><<<gridSize, blockSize>>>(
      current_timestep, 
      domain, core, ext, drho_c, drho_n, dphi_c, dphi_n, 
      *dncps, dcps);
  hipDeviceSynchronize();
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices, kernel function");

  unsigned long long ncps = 0;
  hipMemcpy(&ncps, dncps, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMemcpyDeviceToHost, dncps");
  fprintf(stderr, "ncps=%llu\n", ncps);

  std::vector<cp_t> cps(ncps);
  hipMemcpy(cps.data(), dcps, sizeof(cp_t) * ncps, hipMemcpyDeviceToHost);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMemcpyDeviceToHost");
  
  if (drho_c) hipFree(drho_c);
  if (drho_n) hipFree(drho_n);
  if (dphi_c) hipFree(dphi_c);
  if (dphi_n) hipFree(dphi_n);
  hipFree(dncps);
  hipFree(dcps);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipFree");
 
  hipDeviceSynchronize();
  auto t1 = std::chrono::high_resolution_clock::now();
  float duration = std::chrono::duration_cast<std::chrono::nanoseconds>(t1 - t0).count() * 1e-9;

  fprintf(stderr, "exitting gpu kernel, ncps=%llu, time=%f\n", ncps, duration);
  
  return cps;
}

std::vector<cp_t>
extract_tdgl_vortex_3dt_cuda(
    int scope, 
    int current_timestep, 
    const ftk::lattice& domain,
    const ftk::lattice& core, 
    const ftk::lattice& ext, 
    const double *rho_c, 
    const double *rho_l,
    const double *phi_c, 
    const double *phi_l)
{
  lattice4_t D(domain);
  lattice4_t C(core);
  lattice3_t E(ext);

  if (scope == scope_interval) 
    return extract_tdgl_vortex_3dt<scope_interval>(current_timestep, D, C, E, rho_c, rho_l, phi_c, phi_l);
  if (scope == scope_ordinal) 
    return extract_tdgl_vortex_3dt<scope_ordinal>(current_timestep, D, C, E, rho_c, rho_l, phi_c, phi_l);
  else // scope == 2
    return extract_tdgl_vortex_3dt<scope_all>(current_timestep, D, C, E, rho_c, rho_l, phi_c, phi_l);
}
