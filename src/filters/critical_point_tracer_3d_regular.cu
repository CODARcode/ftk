#include "hip/hip_runtime.h"
#include <nvfunctional>
#include <cstdio>
#include <cassert>
// #include <ftk/filters/critical_point_tracker_2d.hh>
#include <ftk/numeric/inverse_linear_interpolation_solver.hh>
#include <ftk/numeric/linear_interpolation.hh>
#include <ftk/hypermesh/lattice.hh>
#include <ftk/filters/critical_point.hh>
#include "common.cuh"

__device__
bool detect_cp3t(
    const double v[4][3],
    const int vertices[4][4], 
    cp4_t &cp)
{
  double mu[4];
  bool succ = ftk::inverse_lerp_s3v3(v, mu);
 
  if (succ) {
    double X[4][4];
    for (int i = 0; i < 4; i ++)
      for (int j = 0; j < 4; j ++)
        X[i][j] = vertices[i][j];
    ftk::lerp_s3v4(X, mu, cp.x);
    return true;
  } else 
    return false;
}

__device__
bool check_simplex_cp3t_streaming_ordinal(
    int current_timestep,
    const lattice3_t& core, 
    const lattice3_t& ext, 
    const element43_t& e, 
    const double *V, 
    cp4_t &cp)
{
  return false;
}

__device__
bool check_simplex_cp3t_streaming_interval(
    int current_timestep,
    const lattice3_t& core,
    const lattice4_t& ext, 
    const element43_t& e, 
    const double *V0, const double *V1, 
    cp4_t &cp)
{
  return false;
}

template <int scope=0>
__device__
bool check_simplex_cp3t(
    const lattice4_t& core, 
    const lattice4_t& ext, // array dimension
    const element43_t& e, 
    const double *V, // , const double *V1, // vector field for two adjacent timesteps
    cp4_t &cp)
{
  int vertices[4][4];
  for (int i = 0; i < 4; i ++)
    for (int j = 0; j < 4; j ++) {
      vertices[i][j] = e.corner[j]
        + unit_simplices_4_3[e.type][i][j];
      if (vertices[i][j] < core.st[j] || 
          vertices[i][j] > core.st[j] + core.sz[j] - 1)
        return false;
    }

  double v[4][3];
  for (int i = 0; i < 4; i ++) {
    size_t k = ext.to_index(vertices[i]);
    for (int j = 0; j < 3; j ++)
      v[i][j] = V[k*3+j]; // V has three channels
  }

  return detect_cp3t(v, vertices, cp);
}

template <int scope=0>
__global__
void sweep_simplices(
    const lattice4_t core,
    const lattice4_t ext, const double *V, 
    unsigned long long &ncps, cp4_t *cps)
{
  int tid = getGlobalIdx_3D_1D();
  const element43_t e = element43_from_index<scope>(core, tid);

  cp4_t cp;
  bool succ = check_simplex_cp3t<scope>(core, ext, e, V, cp);
  if (succ) {
    unsigned long long i = atomicAdd(&ncps, 1ul);
    cp.tag = tid;
    cps[i] = cp;
  }
}

template <int scope=0>
static std::vector<cp4_t> extract_cp3dt(
    const lattice4_t& core, 
    const lattice4_t& ext, const double *V/* 5D array: 2*W*H*D*T */)
{
  fprintf(stderr, "init GPU...\n");
  const size_t ntasks = core.n() * 60; // ntypes_4[3] = 60; 
  const int maxGridDim = 1024;
  const int blockSize = 256;
  const int nBlocks = idivup(ntasks, blockSize);
  dim3 gridSize;
  
  if (nBlocks >= maxGridDim) 
    gridSize = dim3(idivup(nBlocks, maxGridDim), maxGridDim);
  else 
    gridSize = dim3(nBlocks);

  double *dV;
  hipMalloc((void**)&dV, 3 * sizeof(double) * ext.n());
  hipMemcpy(dV, V, 3 * sizeof(double) * ext.n(), hipMemcpyHostToDevice);

  unsigned long long *dncps; // number of cps
  hipMalloc((void**)&dncps, sizeof(unsigned long long));
  hipMemset(dncps, 0, sizeof(unsigned long long));

  cp4_t *dcps;
  hipMalloc((void**)&dcps, sizeof(cp4_t) * core.n());
  hipDeviceSynchronize();
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMalloc/hipMemcpy");

  fprintf(stderr, "calling kernel func...\n");
  sweep_simplices<scope><<<gridSize, blockSize>>>(core, ext, dV, *dncps, dcps);
  hipDeviceSynchronize();
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices, kernel function");

  unsigned long long ncps = 0;
  hipMemcpy(&ncps, dncps, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMemcpyDeviceToHost, dncps");
  fprintf(stderr, "ncps=%lu\n", ncps);

  std::vector<cp4_t> cps(ncps);
  hipMemcpy(cps.data(), dcps, sizeof(cp4_t) * ncps, hipMemcpyDeviceToHost);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipMemcpyDeviceToHost");
  
  hipFree(dV);
  hipFree(dncps);
  hipFree(dcps);
  checkLastCudaError("[FTK-CUDA] error: sweep_simplices: hipFree");
 
  hipDeviceSynchronize();
  fprintf(stderr, "exit, ncps=%lu\n", ncps);

  return cps;
}

std::vector<cp4_t>
extract_cp3dt_cuda(
    const ftk::lattice& core, int scope,
    const ftk::lattice& ext, const double *V)
{
  lattice4_t C(core), E(ext);

  if (scope == 0) return extract_cp3dt<0>(C, E, V);
  else if (scope == 1) return extract_cp3dt<1>(C, E, V);
  else if (scope == 2) return extract_cp3dt<2>(C, E, V);
  else {
    assert(false);
    return std::vector<cp4_t>(); // make compiler happy
  }
}
